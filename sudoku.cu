#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <fstream>
#include <vector>
#include <math.h>
#include <iomanip>
#include <sys/time.h>
#include <string>

// UNASSIGNED is used for empty cells in sudoku grid
#define UNASSIGNED 0

// N is used for size of Sudoku grid. Size will be NxN
#define N 9

// This function finds an entry in grid that is still unassigned
__host__ __device__ bool FindUnassignedLocation(int grid[N][N], int &row, int &col);

// Checks whether it will be legal to assign num to the given row,col
__host__ __device__ bool isSafe(int grid[N][N], int row, int col, int num);

/* Takes a partially filled-in grid and attempts to assign values to
all unassigned locations in such a way to meet the requirements
for Sudoku solution (non-duplication across rows, columns, and boxes) */
__host__ __device__ bool SolveSudoku(int grid[N][N])
{
// printf("Entered Solve Function \n");
        int row, col;

        // If there is no unassigned location, we are done
        if (!FindUnassignedLocation(grid, row, col))
        {
                return true; // success!
        }

                                         // consider digits 1 to 9
        for (int num = 1; num <= 9; num++)
        {
//      printf("Entered for loop :: num %d :: row %d :: col %d \n",num,row,col);
                // if looks promising
                if (isSafe(grid, row, col, num))
                {
//              printf("Entered If block ::num %d :: row %d :: col %d \n",num,row,col);
                        // make tentative assignment
                        grid[row][col] = num;

                        // return, if success, yay!
                        if (SolveSudoku(grid))
                        {
//                      printf("Entered second If block ::num %d :: row %d :: col %d \n",num,row,col);
                                return true;
                        }

                        // failure, unmake & try again
                        grid[row][col] = UNASSIGNED;
                }
        }
        return false; // this triggers backtracking
}

/* Searches the grid to find an entry that is still unassigned. If
found, the reference parameters row, col will be set the location
that is unassigned, and true is returned. If no unassigned entries
remain, false is returned. */
__host__ __device__ bool FindUnassignedLocation(int grid[N][N], int &row, int &col)
{
        for (row = 0; row < N; row++)
                for (col = 0; col < N; col++)
                        if (grid[row][col] == UNASSIGNED)
                                return true;
        return false;
}

/* Returns a boolean which indicates whether any assigned entry
in the specified row matches the given number. */
__host__ __device__ bool UsedInRow(int grid[N][N], int row, int num)
{
        for (int col = 0; col < N; col++)
                if (grid[row][col] == num)
                        return true;
        return false;
}

/* Returns a boolean which indicates whether any assigned entry
in the specified column matches the given number. */
__host__ __device__ bool UsedInCol(int grid[N][N], int col, int num)
{
        for (int row = 0; row < N; row++)
                if (grid[row][col] == num)
                        return true;
        return false;
}

/* Returns a boolean which indicates whether any assigned entry
within the specified 3x3 box matches the given number. */
__host__ __device__ bool UsedInBox(int grid[N][N], int boxStartRow, int boxStartCol, int num)
{
        for (int row = 0; row < 3; row++)
                for (int col = 0; col < 3; col++)
                        if (grid[row + boxStartRow][col + boxStartCol] == num)
                                return true;
        return false;
}

/* Returns a boolean which indicates whether it will be legal to assign
num to the given row,col location. */
__host__ __device__ bool isSafe(int grid[N][N], int row, int col, int num)
{
        /* Check if 'num' is not already placed in current row,
        current column and current 3x3 box */
        return !UsedInRow(grid, row, num) &&
                !UsedInCol(grid, col, num) &&
                !UsedInBox(grid, row - row % 3, col - col % 3, num);
}

/* A utility function to print grid */
__host__ __device__ void printGrid(int grid[N][N])
{
        for (int row = 0; row < N; row++)
        {
                for (int col = 0; col < N; col++)
                        printf("%2d", grid[row][col]);
                printf("\n");
        }
}

void checkErrors(const char label[])
{
  // we need to synchronise first to catch errors due to
  // asynchroneous operations that would otherwise
  // potentially go unnoticed

  hipError_t err;

  err = hipDeviceSynchronize();
  if (err != hipSuccess)
  {
    char *e = (char*) hipGetErrorString(err);
    fprintf(stderr, "CUDA Error: %s (at %s)", e, label);
  }

  err = hipGetLastError();
  if (err != hipSuccess)
  {
    char *e = (char*) hipGetErrorString(err);
    fprintf(stderr, "CUDA Error: %s (at %s)", e, label);
  }
}

__global__ void sudokuSolve(int matrix[N][N],bool result)
{
//printf(" Entered Global function :: result :: %d \n",result);
int blockRow;
int blockCol;
                //int blockRow = blockIdx.y;
                //int blockCol = blockIdx.x;

                // If there is no unassigned location, we are done
                if (!FindUnassignedLocation(matrix, blockRow, blockCol))
                {
                        result = true; // success!
                        return;
                }

                                                 // consider digits 1 to 9
                for (int num = 1; num <= 9; num++)
                {
//              printf(" Entered For Loop in Global :: num %d :: row %d ::col %d \n",num,blockRow,blockCol);
                        // if looks promising
                        if (isSafe(matrix, blockRow, blockCol, num))
                        {
//                      printf(" Entered If Block in Global \n");
                                // make tentative assignment
                                matrix[blockRow][blockCol] = num;

                                // return, if success, yay!
                                if (SolveSudoku(matrix))
                                {
                                        result = true;
                                        return;
                                }

                                // failure, unmake & try again
                                matrix[blockRow][blockCol] = UNASSIGNED;


                //int blockCol = blockIdx.x;

                // If there is no unassigned location, we are done
                if (!FindUnassignedLocation(matrix, blockRow, blockCol))
                {
                        result = true; // success!
                        return;
                }

                                                 // consider digits 1 to 9
                for (int num = 1; num <= 9; num++)
                {
//              printf(" Entered For Loop in Global :: num %d :: row %d ::col %d \n",num,blockRow,blockCol);
                        // if looks promising
                        if (isSafe(matrix, blockRow, blockCol, num))
                        {
//                      printf(" Entered If Block in Global \n");
                                // make tentative assignment
                                matrix[blockRow][blockCol] = num;

                                // return, if success, yay!
                                if (SolveSudoku(matrix))
                                {
                                        result = true;
                                        return;
                                }

                                // failure, unmake & try again
                                matrix[blockRow][blockCol] = UNASSIGNED;
                        }
                }
                result = false; // this triggers backtracking // maybe
                return;

}

// BEGIN: timing and error checking routines (do not modify)

// Returns the current time in microseconds
long long start_timer() {
        struct timeval tv;
        gettimeofday(&tv, NULL);
        return tv.tv_sec * 1000000 + tv.tv_usec;
}


// Prints the time elapsed since the specified time
long long stop_timer(long long start_time, std::string name) {
        struct timeval tv;
        gettimeofday(&tv, NULL);
        long long end_time = tv.tv_sec * 1000000 + tv.tv_usec;
        std::cout << std::setprecision(5);
        std::cout << name << ": " << ((float) (end_time - start_time)) / (1000 * 1000) << " sec\n";
        return end_time - start_time;
}


/* Driver Program to test above functions */
int main()
{
        // 0 means unassigned cells
        int grid[N][N] = {
        { 3, 0, 6, 5, 0, 8, 4, 0, 0 },
        { 5, 2, 0, 0, 0, 0, 0, 0, 0 },
        { 0, 8, 7, 0, 0, 0, 0, 3, 1 },
        { 0, 0, 3, 0, 1, 0, 0, 8, 0 },
        { 9, 0, 0, 8, 6, 3, 0, 0, 5 },
        { 0, 5, 0, 0, 9, 0, 6, 0, 0 },
        { 1, 3, 0, 0, 0, 0, 2, 5, 0 },
        { 0, 0, 0, 0, 0, 0, 0, 7, 4 },
        { 0, 0, 5, 2, 0, 6, 3, 0, 0 } };

        // coppyof first to use for parallel because serial changes original data
        int grid2[N][N] = {
        { 3, 0, 6, 5, 0, 8, 4, 0, 0 },
        { 5, 2, 0, 0, 0, 0, 0, 0, 0 },
        { 0, 8, 7, 0, 0, 0, 0, 3, 1 },
        { 0, 0, 3, 0, 1, 0, 0, 8, 0 },
        { 9, 0, 0, 8, 6, 3, 0, 0, 5 },
        { 0, 5, 0, 0, 9, 0, 6, 0, 0 },
        { 1, 3, 0, 0, 0, 0, 2, 5, 0 },
        { 0, 0, 0, 0, 0, 0, 0, 7, 4 },
        { 0, 0, 5, 2, 0, 6, 3, 0, 0 } };

        printf("***********Input Puzzle*********** \n");
        printGrid(grid);
long long CPU_start_time = start_timer();
        if (SolveSudoku(grid) == true)
        {
                printf("*******Serial Solved Puzzle******* \n");
                printGrid(grid);
        }
        else
                printf("No solution exists");
 long long CPU_time = stop_timer(CPU_start_time, "CPU Run Time");

///////////////////////////////////////// Cuda ////////////////////////////////

        // set up variables
        int d_matrix[N][N];
        bool *cudaresult;
        bool *d_result;
long long GPU_start_time = start_timer();

        //set up gpu memory
        hipMalloc((void**)&d_matrix, (1*sizeof(int)));
        hipMalloc((void**)&d_result, (1*sizeof(bool)));
        //checkErrors("The Mallocs \n");

        // put the data into gpu memory
        hipMemcpy(d_matrix, grid, (1*sizeof(int)), hipMemcpyHostToDevice);
        //checkErrors("The Memcopy 1-1 matrix \n");
        hipMemcpy(d_result, cudaresult, (1*sizeof(bool)), hipMemcpyHostToDevice);
        //checkErrors("The Memcopys 1-2 result \n");

        // run the kernal
        sudokuSolve<< <1, 1>> >(d_matrix,d_result);
        //checkErrors("The kernal \n");

        // copy back memory from GPU to CPU
        hipMemcpy(grid, d_matrix, (81*sizeof(int)), hipMemcpyDeviceToHost);
        //checkErrors("The Memcopys 2-1 matrix \n");
        hipMemcpy(cudaresult, d_result, (1*sizeof(bool)), hipMemcpyDeviceToHost);
        //checkErrors("The Memcopys 2-2 result \n");


        // print out parellel solved puzzle
        printf("********CUDA Solved Puzzle******** \n");
        printGrid(grid);

 long long GPU_starttime_endtimer = stop_timer(GPU_start_time, "GPU Total run time ");
        hipFree(d_matrix);
        hipFree(d_result);


        return 0;
}
